#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "math.hpp"
#include "types.hpp"
#include "grid_stride_range.hpp"
#include "execution.hpp"

#include "../cuda4dnn/csl/stream.hpp"
#include "../cuda4dnn/csl/tensor.hpp"
#include "../cuda4dnn/csl/span.hpp"

#include <hip/hip_runtime.h>

using namespace cv::dnn::cuda4dnn::csl;
using namespace cv::dnn::cuda4dnn::csl::device;

namespace cv { namespace dnn { namespace cuda4dnn { namespace kernels {

    namespace raw {
        template <class T>
        __global__ void resize_nn(
            Span<T> output, size_type out_height, size_type out_width,
            View<T> input, size_type in_height, size_type in_width)
        {
            auto in_image_size = in_height * in_width;
            auto out_image_size = out_height * out_width;

            /* o2i = output to input */
            auto o2i_fx = static_cast<float>(in_width) / out_width;
            auto o2i_fy = static_cast<float>(in_height) / out_height;

            /* think of the output and input as a collection of 2d images with the last axis
             * representing the width and the last but one axis representing the height
             *
             * the remaining axis together form a collection of these images
             */
            for (auto idx : grid_stride_range(output.size())) {
                const index_type n = idx / out_image_size;
                const index_type x = (idx % out_image_size) % out_width;
                const index_type y = (idx % out_image_size) / out_width;

                auto in_x = static_cast<index_type>(x * o2i_fx);
                auto in_y = static_cast<index_type>(y * o2i_fy);

                index_type in_idx = n * in_image_size + in_y * in_width + in_x;
                output[idx] = input[in_idx];
            }
        }

        template <class T>
        __global__ void resize_bilinear(
            Span<T> output, size_type out_height, size_type out_width,
            View<T> input, size_type in_height, size_type in_width,
            float o2i_fy, float o2i_fx)
        {
            auto in_image_size = in_height * in_width;
            auto out_image_size = out_height * out_width;

            /* think of the output and input as a collection of 2d images with the last axis
             * representing the width and the last but one axis representing the height
             *
             * the remaining axis together form a collection of these images
             */
            for (auto idx : grid_stride_range(output.size())) {
                const index_type n = idx / out_image_size;
                const index_type x = (idx % out_image_size) % out_width;
                const index_type y = (idx % out_image_size) / out_width;

                auto in_x = x * o2i_fx;
                auto in_y = y * o2i_fy;

                auto in_x0 = static_cast<index_type>(in_x);
                auto in_y0 = static_cast<index_type>(in_y);

                using device::min;
                auto in_x1 = min<index_type>(in_x0 + 1, in_width - 1);
                auto in_y1 = min<index_type>(in_y0 + 1, in_height - 1);

                const index_type in_offset_r0 = n * in_image_size + in_y0 * in_width;
                const index_type in_offset_r1 = n * in_image_size + in_y1 * in_width;

                auto v_00 = input[in_offset_r0 + in_x0],
                     v_01 = input[in_offset_r0 + in_x1],
                     v_10 = input[in_offset_r1 + in_x0],
                     v_11 = input[in_offset_r1 + in_x1];

                output[idx] =
                    v_00 +
                    T(in_y - in_y0) * T(v_10 - v_00) +
                    T(in_x - in_x0) * T(v_01 - v_00) +
                    T(in_y - in_y0) * T(in_x - in_x0) * T(v_11 - v_01 - v_10 + v_00);
            }
        }
    }

    template <class T>
    void resize_nn(const Stream& stream, TensorSpan<T> output, TensorView<T> input) {
        auto in_height = input.get_axis_size(-2);
        auto in_width = input.get_axis_size(-1);

        auto out_height = output.get_axis_size(-2);
        auto out_width = output.get_axis_size(-1);

        auto kernel = raw::resize_nn<T>;
        auto policy = make_policy(kernel, output.size(), 0, stream);
        launch_kernel(kernel, policy, output, out_height, out_width, input, in_height, in_width);
    }

    template void resize_nn<__half>(const Stream&, TensorSpan<__half>, TensorView<__half>);
    template void resize_nn<float>(const Stream&, TensorSpan<float>, TensorView<float>);

    template <class T>
    void resize_bilinear(const Stream& stream, TensorSpan<T> output, TensorView<T> input, float scale_y, float scale_x) {
        auto in_height = input.get_axis_size(-2);
        auto in_width = input.get_axis_size(-1);

        auto out_height = output.get_axis_size(-2);
        auto out_width = output.get_axis_size(-1);

        auto kernel = raw::resize_bilinear<T>;
        auto policy = make_policy(kernel, output.size(), 0, stream);
        launch_kernel(kernel, policy, output, out_height, out_width, input, in_height, in_width, scale_y, scale_x);
    }

    template void resize_bilinear<__half>(const Stream&, TensorSpan<__half>, TensorView<__half>, float, float);
    template void resize_bilinear<float>(const Stream&, TensorSpan<float>, TensorView<float>, float, float);

}}}} /* namespace cv::dnn::cuda4dnn::kernels */
